#include "hip/hip_runtime.h"
#include <stdio.h>
// CUDA header file
#include "hip/hip_runtime.h"
#include <>
#ifndef __HIPCC__
#define __HIPCC__
#endif
// UNASSIGNED is used for empty cells in Sudoku grid 
#define UNASSIGNED 0

// BOX_W is used for the length of one of the square sub-regions of the Sudoku grid.
// Overall length will be N * N.
#define BOX_W 4
#define N (BOX_W * BOX_W)
#define NSQRD  (N * N)
#define N_2 ( N * N  * 2)
#define CANS 1


// Function to check if a number can be placed in a cell
int is_valid(int board[N][N], int row, int col, int num, int BLOCK_SIZE) {
    // Check row and column
    for (int i = 0; i < N; i++) {
        if (board[row][i] == num || board[i][col] == num) {
            return 0;
        }
    }

    // Check subgrid
    int startRow = (row / BLOCK_SIZE) * BLOCK_SIZE;
    int startCol = (col / BLOCK_SIZE) * BLOCK_SIZE;
    for (int i = 0; i < BLOCK_SIZE; i++) {
        for (int j = 0; j < BLOCK_SIZE; j++) {
            if (board[startRow + i][startCol + j] == num) {
                return 0;
            }
        }
    }

    return 1;
}

// Backtracking solver
int solve_sudoku_serial(int board[N][N], int BLOCK_SIZE) {
    for (int row = 0; row < N; row++) {
        for (int col = 0; col < N; col++) {
            if (board[row][col] == 0) {
                for (int num = 1; num <= N; num++) {
                    if (is_valid(board, row, col, num, BLOCK_SIZE)) {
                        board[row][col] = num;

                        if (solve_sudoku_serial(board, BLOCK_SIZE)) {
                            return 1;
                        }

                        board[row][col] = 0; // Backtrack
                    }
                }
                return 0; // No valid number found
            }
        }
    }
    return 1; // Sudoku solved
}

__global__ void solve(int* d_a, int* d_flag) {
	// Used to remember which row | col | box ( section ) have which values
	__shared__ bool rowHas[N][N];
	__shared__ bool colHas[N][N];
	__shared__ bool boxHas[N][N];

	// Used to ensure that the table has changed
	__shared__ bool changed;

	// Number of spaces which can place the number in each section
	__shared__ int rowCount[N][N];
	__shared__ int colCount[N][N];
	__shared__ int boxCount[N][N];

	__shared__ int sudoku[NSQRD];
	// Where the square is located in the Sudoku
	int row = threadIdx.x;
	int col = threadIdx.y;
	int box = col / BOX_W + (row / BOX_W) * BOX_W;

	// Square's location in the Sudoku
	int gridIdx = row * N + col;
	int Ngrid = N * N + gridIdx;

	sudoku[gridIdx] = d_a[gridIdx] - 1;
	// Unique identifier for each square in row, col, box
	// Corresponds to the generic Sudoku Solve
	// Using a Sudoku to solve a Sudoku !!!
	int offset = row + (col % BOX_W) * BOX_W + (box % BOX_W);

	bool notSeen[N];
	for (int i = 0; i < N; ++i)
		notSeen[i] = true;

	rowHas[row][col] = false;
	colHas[row][col] = false;
	boxHas[row][col] = false;
	__syncthreads(); 

	// Previous loop has not changed any values
	int loopCount = 0;
	do {
		// RESET counters
		rowCount[row][col] = 0;
		colCount[row][col] = 0;
		boxCount[row][col] = 0;

		int s_at = sudoku[gridIdx];
		if (s_at != -1) {
			rowHas[row][s_at] = true;
			colHas[col][s_at] = true;
			boxHas[box][s_at] = true;
		}
		

		__syncthreads();

		changed = false;
		int count = 0;  // number of values which can fit in this square
		int guess = 0; // last value found which can fit in this square
		for (int idx = 0; idx < N; ++idx) {
			// Ensures that every square in each section is working on a different number in the section
			int num = (idx + offset) % N;
			if (s_at == -1 && notSeen[num]) {
				if (rowHas[row][num] || boxHas[box][num] || colHas[col][num])
					notSeen[num] = false;
				else {
					++count;
					guess = num;
					rowCount[row][num] += Ngrid;
					colCount[col][num] += Ngrid;
					boxCount[box][num] += Ngrid;
				}
			}
			__syncthreads();
		}
		if (count == 1)
			sudoku[gridIdx] = guess;

		// Find values which can go in only one spot in the section
		guess = rowCount[row][col];
		if (guess > 0 && guess < N_2)
			sudoku[guess - NSQRD] = col;
		
		guess = boxCount[row][col];
		if (guess > 0 && guess < N_2)
			sudoku[guess - NSQRD] = col;
		
		guess = colCount[row][col];
		if (guess > 0 && guess < N_2)
			sudoku[guess - NSQRD] = col;

		if (sudoku[gridIdx] == -1)
			changed = true;
		__syncthreads();

	} while (changed && ++loopCount < N);

	if (row == 0 && col == 0) {
		// print the number of loops
		printf("Number of loops: %d\n", loopCount);
		}

	
	int s_at = sudoku[gridIdx];
	if (s_at != -1) {
		rowHas[row][s_at] = true;
		colHas[col][s_at] = true;
		boxHas[box][s_at] = true;
	}

	d_a[gridIdx] = sudoku[gridIdx] + 1;
	__syncthreads();
	if (!(rowHas[row][col] && colHas[row][col] && boxHas[row][col]))
		*d_flag = 1; //HAVE NOT SOLVED THE SUDOKU
		
}


// read file
int readMatrixFromFile(const char* filename, int matrix[N][N]) {
    FILE* file = fopen(filename, "r");
    if (!file) {
        printf("Error: Unable to open file %s\n", filename);
        return -1;  // fail
    }

    // obtain the data
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (fscanf(file, "%d", &matrix[i][j]) != 1) {
                printf("Error: Invalid matrix data at row %d, column %d\n", i + 1, j + 1);
                fclose(file);
                return -1;  // fail
            }
        }
    }

    fclose(file);
    return 0;  // success
}

void print(int result[N][N]) {
	for (int row = 0; row < N; row++) {
		for (int col = 0; col < N; col++)
			printf("%3d", result[row][col]);
		printf("\n");
	}
}

int main() {
    // List of file names
    const char* filenames[] = {
        "testcase/16_easy.txt",
        "testcase/16_medium.txt",
        "testcase/16_hard.txt"
    };
    int num_files = sizeof(filenames) / sizeof(filenames[0]);

    for (int file_idx = 0; file_idx < num_files; file_idx++) {
        const char* filename = filenames[file_idx];
        int h_a[N][N];

        // read matrix from file
        if (readMatrixFromFile(filename, h_a)) {
            printf("Failed to read matrix from file %s.\n", filename);
            continue; // Skip to the next file
        }

        // print matrix
        printf("Matrix read from file %s:\n", filename);
        print(h_a);

        int* d_a;      // Table
        int* d_flag;   // Flag to indicate if the Sudoku is solved
        int h_flag = 0; // Host flag, initialized to 1 (solved)
        hipMalloc((void**)&d_a, N * N * sizeof(int));
        hipMalloc((void**)&d_flag, sizeof(int));
        // Copy Sudoku to device
        hipMemcpy(d_a, h_a, N * N * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_flag, &h_flag, sizeof(int), hipMemcpyHostToDevice);

        // Set up the grid and block dimensions
        dim3 dBlock(N, N);

        // record start time
        hipEvent_t start, stop;
        float elapsedTime;
		double time_taken = 0.;

        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        solve<<<1, dBlock>>>(d_a, d_flag);

        // record end time
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);

        // printf("Solve function execution time for file %s: %f ms\n", filename, elapsedTime);

        // Copy Sudoku and flag back to host
        hipMemcpy(h_a, d_a, N * N * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(&h_flag, d_flag, sizeof(int), hipMemcpyDeviceToHost);

        // if cuda is not able to solve the sudoku, let the remaining solve by serial
        if (h_flag == 1) {
            // serial solve
			print(h_a);

			printf("Solving the remaining Sudoku by serial...\n");
            clock_t c_start = clock();
            solve_sudoku_serial(h_a, BOX_W);
            clock_t c_end = clock();
            time_taken = ((double)(c_end - c_start)) / CLOCKS_PER_SEC;
            // printf("Serial solve execution time for file %s: %f ms\n", filename, time_taken * 1000);
        }

        // Check if solved
        if (h_a[0][0])
            print(h_a);
        else
            printf("No solution could be found for file %s.\n", filename);

        hipFree(d_a);
        hipFree(d_flag);
		
		// print total time elapsedTime + time_taken
		printf("Total execution time for file %s: %f ms\n", filename, elapsedTime+time_taken * 1000);
    }

    return 0;
}